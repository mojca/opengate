#include "hip/hip_runtime.h"
#include "GateGPUParticle.hh"
#include "GateToGPUImageSPECT.hh"

__device__ float vector_dot(float3 u, float3 v) {
    return u.x*v.x + u.y*v.y + u.z*v.z;
}

__device__ float3 vector_sub(float3 u, float3 v) {
    return make_float3(u.x-v.x, u.y-v.y, u.z-v.z);
}

__device__ float3 vector_add(float3 u, float3 v) {
    return make_float3(u.x+v.x, u.y+v.y, u.z+v.z);
}

__device__ float3 vector_mag(float3 u, float a) {
    return make_float3(u.x*a, u.y*a, u.z*a);
}

__device__ unsigned int binary_search(float position, float *tab, unsigned int maxid ) {

    unsigned short int begIdx = 0;
    unsigned short int endIdx = maxid - 1;
    unsigned short int medIdx = endIdx / 2;

    while (endIdx-begIdx > 1) {
        if (position < tab[medIdx]) {begIdx = medIdx;}
        else {endIdx = medIdx;}
        medIdx = (begIdx+endIdx) / 2;
    }
    return medIdx;
}

__global__ void kernel_map_entry(float *d_px, float *d_py, float *d_pz,
                                 float *d_entry_collim_y, float *d_entry_collim_z,
                                 int *d_hole, unsigned int y_size, unsigned int z_size,
                                 int particle_size) {

    unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (id >= particle_size) {return;}
    if( d_py[ id ] > d_entry_collim_y[ 0 ] || d_py[ id ] < d_entry_collim_y[ y_size - 1 ] )
    {
        d_hole[ id ]=-1;
        return;
    }
		if( d_pz[ id ] > d_entry_collim_z[ 0 ] || d_pz[ id ] < d_entry_collim_z[ z_size - 1 ] )
    {
        d_hole[ id ] = -1;
        return;
    }

    unsigned int index_entry_y = binary_search( d_py[ id ], d_entry_collim_y, y_size );
    unsigned int index_entry_z = binary_search( d_pz[ id ], d_entry_collim_z, z_size );

    unsigned char is_in_hole_y = ( index_entry_y & 1 ) ? 0 : 1;
    unsigned char is_in_hole_z = ( index_entry_z & 1 ) ? 0 : 1;

    unsigned char in_hole = is_in_hole_y & is_in_hole_z;

    d_hole[ id ] = ( in_hole )? index_entry_y * z_size + index_entry_z : -1;
}

__global__ void kernel_map_projection(float *d_px, float *d_py, float *d_pz,
                                      float *d_dx, float *d_dy, float *d_dz,
                                      int *d_hole, float planeToProject,
                                      unsigned int particle_size) {

    unsigned int id = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x;
    if( id >= particle_size ) return;
    if( d_hole[ id ] == -1 ) return;

    float3 n  = make_float3( -1.0f, 0.0f, 0.0f );
    float3 v0 = make_float3( planeToProject, 0.0f, 0.0f );
    float3 d  = make_float3( d_dx[ id ], d_dy[ id ], d_dz[ id ] );
    float3 p  = make_float3( d_px[ id ], d_py[ id ], d_pz[ id ] );

    float s = __fdividef( vector_dot( n, vector_sub( v0, p ) ), vector_dot( n, d ) );
    float3 newp = vector_add( p, vector_mag( d, s ) );

    d_px[id] = newp.x;
    d_py[id] = newp.y;
    d_pz[id] = newp.z;
}


__global__ void kernel_map_exit(float *d_px, float *d_py, float *d_pz,
                                float *d_exit_collim_y, float *d_exit_collim_z,
                                int *d_hole, unsigned int y_size, unsigned int z_size,
                                int particle_size) {

    unsigned int id = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x;
    if( id >= particle_size ) return;
    if( d_hole[ id ] == -1 ) return;

    if( d_py[ id ] > d_exit_collim_y[ 0 ] || d_py[ id ] < d_exit_collim_y[ y_size - 1 ] )
    {
        d_hole[ id ]=-1;
        return;
    }
    if( d_pz[ id ] > d_exit_collim_z[ 0 ] || d_pz[ id ] < d_exit_collim_z[ z_size - 1 ] )
    {
        d_hole[ id ] = -1;
        return;
    }

    unsigned int index_exit_y = binary_search( d_py[ id ], d_exit_collim_y, y_size );
    unsigned int index_exit_z = binary_search( d_pz[ id ], d_exit_collim_z, z_size );

    unsigned char is_in_hole_y = ( index_exit_y & 1 )? 0 : 1;
    unsigned char is_in_hole_z = ( index_exit_z & 1 )? 0 : 1;

    unsigned char in_hole = is_in_hole_y & is_in_hole_z;

    int newhole = ( in_hole )? index_exit_y * z_size + index_exit_z : -1;

    if( newhole == -1 )
    {
        d_hole[ id ] = -1;
        return;
    }

    if( newhole != d_hole[ id ] )
    {
        d_hole[ id ] = -1;
    }
}



void GateGPUCollimator_init(GateGPUCollimator *collimator) {

    hipSetDevice(collimator->cudaDeviceID);

    unsigned int y_size = collimator->y_size;
    unsigned int z_size = collimator->z_size;

    unsigned int mem_float_y = y_size * sizeof(float);
    unsigned int mem_float_z = z_size * sizeof(float);


    float* d_entry_collim_y;
    float* d_entry_collim_z;
    float* d_exit_collim_y;
    float* d_exit_collim_z;

    hipMalloc((void**) &d_entry_collim_y, mem_float_y);
    hipMalloc((void**) &d_entry_collim_z, mem_float_z);
    hipMalloc((void**) &d_exit_collim_y, mem_float_y);
    hipMalloc((void**) &d_exit_collim_z, mem_float_z);

    hipMemcpy(d_entry_collim_y, collimator->entry_collim_y, mem_float_y, hipMemcpyHostToDevice);
    hipMemcpy(d_entry_collim_z, collimator->entry_collim_z, mem_float_z, hipMemcpyHostToDevice);
    hipMemcpy(d_exit_collim_y, collimator->exit_collim_y, mem_float_y, hipMemcpyHostToDevice);
    hipMemcpy(d_exit_collim_z, collimator->exit_collim_z, mem_float_z, hipMemcpyHostToDevice);

    collimator->gpu_entry_collim_y = d_entry_collim_y;
    collimator->gpu_entry_collim_z = d_entry_collim_z;
    collimator->gpu_exit_collim_y = d_exit_collim_y;
    collimator->gpu_exit_collim_z = d_exit_collim_z;
}

void GateGPUCollimator_process(GateGPUCollimator *collimator, GateGPUParticle *particle) {

    hipSetDevice(collimator->cudaDeviceID);

    // Read collimator geometry
    float* d_entry_collim_y = collimator->gpu_entry_collim_y;
    float* d_entry_collim_z = collimator->gpu_entry_collim_z;
    float* d_exit_collim_y  = collimator->gpu_exit_collim_y;
    float* d_exit_collim_z  = collimator->gpu_exit_collim_z;
    unsigned int y_size     = collimator->y_size;
    unsigned int z_size     = collimator->z_size;
    float planeToProject    = collimator->planeToProject + particle->px[0];

    // Particles allocation to the Device
    int particle_size = particle-> size;
    unsigned int mem_float_particle = particle_size * sizeof(float);
    unsigned int mem_int_hole = particle_size * sizeof(int);
    float *d_px, *d_py, *d_pz;
    float *d_dx, *d_dy, *d_dz;
    int *d_hole;
    hipMalloc((void**) &d_px, mem_float_particle);
    hipMalloc((void**) &d_py, mem_float_particle);
    hipMalloc((void**) &d_pz, mem_float_particle);
    hipMalloc((void**) &d_dx, mem_float_particle);
    hipMalloc((void**) &d_dy, mem_float_particle);
    hipMalloc((void**) &d_dz, mem_float_particle);
    hipMalloc((void**) &d_hole, mem_int_hole);

    // Array of holes :)
    int *h_hole = (int*)malloc(mem_int_hole);

    // Copy particles from host to device
    hipMemcpy(d_px, particle->px, mem_float_particle, hipMemcpyHostToDevice);
    hipMemcpy(d_py, particle->py, mem_float_particle, hipMemcpyHostToDevice);
    hipMemcpy(d_pz, particle->pz, mem_float_particle, hipMemcpyHostToDevice);
    hipMemcpy(d_dx, particle->dx, mem_float_particle, hipMemcpyHostToDevice);
    hipMemcpy(d_dy, particle->dy, mem_float_particle, hipMemcpyHostToDevice);
    hipMemcpy(d_dz, particle->dz, mem_float_particle, hipMemcpyHostToDevice);

    // Kernel vars
    dim3 threads, grid;
    int block_size = 512;
    int grid_size = (particle_size + block_size - 1) / block_size;
    threads.x = block_size;
    grid.x = grid_size;

    // Kernel map entry
    kernel_map_entry<<<grid, threads>>>(d_px, d_py, d_pz,
                                        d_entry_collim_y, d_entry_collim_z,
                                        d_hole, y_size, z_size,
                                        particle_size);

    // Kernel projection
    kernel_map_projection<<<grid, threads>>>(d_px, d_py, d_pz,
                                             d_dx, d_dy, d_dz,
                                             d_hole, planeToProject, particle_size);

    // Kernel map_exit
    kernel_map_exit<<<grid, threads>>>(d_px, d_py, d_pz,
                                       d_exit_collim_y, d_exit_collim_z,
                                       d_hole, y_size, z_size,
                                       particle_size);

    // Copy particles from device to host
    hipMemcpy(particle->px, d_px, mem_float_particle, hipMemcpyDeviceToHost);
    hipMemcpy(particle->py, d_py, mem_float_particle, hipMemcpyDeviceToHost);
    hipMemcpy(particle->pz, d_pz, mem_float_particle, hipMemcpyDeviceToHost);
    hipMemcpy(h_hole, d_hole, mem_int_hole, hipMemcpyDeviceToHost);

    // Pack data to CPU
    int c = 0;
    int i = 0;
    while( i < particle_size )
    {
        if( h_hole[ i ] == -1 )
        {
            ++i;
            continue;
        }

        //h_hole[ c ] = h_hole[ i ];
        particle->px[ c ] = particle->px[ i ];
        particle->py[ c ] = particle->py[ i ];
        particle->pz[ c ] = particle->pz[ i ];
        particle->dx[ c ] = particle->dx[ i ];
        particle->dy[ c ] = particle->dy[ i ];
        particle->dz[ c ] = particle->dz[ i ];
				particle->eventID[ c ] = particle->eventID[ i ];
				particle->parentID[ c ] = particle->parentID[ i ];
				particle->trackID[ c ] = particle->trackID[ i ];
				particle->t[ c ] = particle->t[ i ];
				particle->E[ c ] = particle->E[ i ];
				particle->type[ c ] = particle->type[ i ];
        ++c;
        ++i;
    }

    particle->size = c;

    // Free memory
    hipFree(d_px);
    hipFree(d_py);
    hipFree(d_pz);
    hipFree(d_dx);
    hipFree(d_dy);
    hipFree(d_dz);
    hipFree(d_hole);
    free(h_hole);
}
