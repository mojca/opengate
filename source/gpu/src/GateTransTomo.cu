#include "hip/hip_runtime.h"
#include "GateGPUIO.hh"
#include <vector>

#define EPS 1.0e-03f
void GPU_GateTransTomo(const GateGPUIO_Input * input, GateGPUIO_Output * output) {

    // FIXME
    // track event ID
    printf("====> GPU START\n");

    // TIMING
    double t_init = time();
    double t_g = time();

    // Select a GPU
    hipSetDevice(input->cudaDeviceID);

    // Seed management
    srand(input->seed);

    // Vars
    int nb_of_particles = input->particles.size();

    // Photons Stacks
    StackParticle photons_d;
    stack_device_malloc(photons_d, nb_of_particles);
    StackParticle photons_h;
    stack_host_malloc(photons_h, nb_of_particles);
    printf(" :: Stack init\n");

    // Materials def, alloc & loading
    Materials materials_h;
    materials_host_malloc(materials_h, input->nb_materials, input->nb_elements_total);

    materials_h.nb_elements = input->mat_nb_elements;
    materials_h.index = input->mat_index;
    materials_h.mixture = input->mat_mixture;
    materials_h.atom_num_dens = input->mat_atom_num_dens;
    materials_h.nb_atoms_per_vol = input->mat_nb_atoms_per_vol;
    materials_h.nb_electrons_per_vol = input->mat_nb_electrons_per_vol;
    materials_h.electron_cut_energy = input->electron_cut_energy;
    materials_h.electron_max_energy = input->electron_max_energy;
    materials_h.electron_mean_excitation_energy = input->electron_mean_excitation_energy;
    materials_h.fX0 = input->fX0;
    materials_h.fX1 = input->fX1;
    materials_h.fD0 = input->fD0;
    materials_h.fC = input->fC;
    materials_h.fA = input->fA;
    materials_h.fM = input->fM;

    Materials materials_d;
    materials_device_malloc(materials_d, input->nb_materials, input->nb_elements_total);
    materials_copy_host2device(materials_h, materials_d);
    printf(" :: Materials init\n");

    // Phantoms
    Volume phantom_d;
    phantom_d.size_in_mm = make_float3(input->phantom_size_x*input->phantom_spacing_x,
                     input->phantom_size_y*input->phantom_spacing_y,
                     input->phantom_size_z*input->phantom_spacing_z);
    phantom_d.voxel_size = make_float3(input->phantom_spacing_x,
                     input->phantom_spacing_y,
                     input->phantom_spacing_z);
    phantom_d.size_in_vox = make_int3(input->phantom_size_x,
                    input->phantom_size_y,
                    input->phantom_size_z);
    phantom_d.nb_voxel_slice = phantom_d.size_in_vox.x * phantom_d.size_in_vox.y;
    phantom_d.nb_voxel_volume = phantom_d.nb_voxel_slice * phantom_d.size_in_vox.z;
    phantom_d.mem_data = phantom_d.nb_voxel_volume * sizeof(unsigned short int);
    volume_device_malloc(phantom_d, phantom_d.nb_voxel_volume);
    hipMemcpy(phantom_d.data, &(input->phantom_material_data[0]), phantom_d.mem_data, hipMemcpyHostToDevice);

    // TIMING
    t_init = time() - t_init;
    double t_in = time();

    // Fill photons stack with particles from GATE
    int i = 0;
    GateGPUIO_Input::ParticlesList::const_iterator iter = input->particles.begin();
    while (iter != input->particles.end()) {
        GateGPUIO_Particle p = *iter;
        photons_h.E[i] = p.E;
        photons_h.dx[i] = p.dx;
        photons_h.dy[i] = p.dy;
        photons_h.dz[i] = p.dz;
        // FIXME need to change the world frame
        p.px += (phantom_d.size_in_mm.x*0.5f);
        p.py += (phantom_d.size_in_mm.y*0.5f);
        p.pz += (phantom_d.size_in_mm.z*0.5f);
        // If the particle in just on the volume boundary, push it inside the volume
        //  (this fix a bug during the navigation)
        if (p.px == phantom_d.size_in_mm.x) p.px -= EPS;
        if (p.py == phantom_d.size_in_mm.y) p.py -= EPS;
        if (p.pz == phantom_d.size_in_mm.z) p.pz -= EPS;
        photons_h.px[i] = p.px;
        photons_h.py[i] = p.py;
        photons_h.pz[i] = p.pz;
        photons_h.t[i] = p.t;
        photons_h.eventID[i] = p.eventID;
        photons_h.trackID[i] = p.trackID;
        photons_h.type[i] = p.type; // FIXME
        photons_h.seed[i] = rand();
        photons_h.endsimu[i] = 0;
        photons_h.active[i] = 1;

        ++iter;
        ++i;
    }
    printf(" :: Load particles from GATE\n");

    // Copy particles from host to device
    stack_copy_host2device(photons_h, photons_d);

    // TIMING
    t_in = time() - t_in;
    double t_init_2 = time();

    // Kernel vars
    dim3 threads, grid;
    int block_size = 512;
    int grid_size = (nb_of_particles + block_size - 1) / block_size;
    threads.x = block_size;
    grid.x = grid_size;

    // Init random
    kernel_brent_init<<<grid, threads>>>(photons_d);


    // Count simulated photons
    int* count_d;
    int count_h = 0;
    hipMalloc((void**) &count_d, sizeof(int));
    hipMemcpy(count_d, &count_h, sizeof(int), hipMemcpyHostToDevice);

    // TIMING
    t_init_2 = time() - t_init_2;
    double t_track = time();

    // Simulation loop
    printf("Before loop\n");
    int step=0;
    while (count_h < nb_of_particles) {
        ++step;
        // Regular navigator
        kernel_NavRegularPhan_Photon_NoSec<<<grid, threads>>>(photons_d, phantom_d,
                                                              materials_d, count_d);

        // get back the number of simulated photons
        hipMemcpy(&count_h, count_d, sizeof(int), hipMemcpyDeviceToHost);

        printf("sim %i %i / %i tot\n", step, count_h, nb_of_particles);
    }

    // TIMING
    t_track = time() - t_track;
    double t_out = time();

    // Copy photons from device to host
    stack_copy_device2host(photons_d, photons_h);

    i=0;
    while (i<nb_of_particles) {

        // Test if the particle was absorbed -> no output.
        if (photons_h.active[i]) {
            GateGPUIO_Particle particle;
            particle.E =  photons_h.E[i];
            particle.dx = photons_h.dx[i];
            particle.dy = photons_h.dy[i];
            particle.dz = photons_h.dz[i];
            particle.px = photons_h.px[i] - (phantom_d.size_in_mm.x*0.5f);
            particle.py = photons_h.py[i] - (phantom_d.size_in_mm.y*0.5f);
            particle.pz = photons_h.pz[i] - (phantom_d.size_in_mm.z*0.5f);
            particle.t =  photons_h.t[i];
            particle.type = photons_h.type[i];
            particle.eventID = photons_h.eventID[i];
            particle.trackID = photons_h.trackID[i];

            output->particles.push_back(particle);
        }
        ++i;
    }

    // TIMING
    t_out = time() - t_out;

    stack_device_free(photons_d);
    stack_host_free(photons_h);
    materials_device_free(materials_d);
    volume_device_free(phantom_d);
    hipFree(count_d);

    t_g = time() - t_g;
    printf(">> GPU: init %e input %e track %e output %e tot %e\n", t_init+t_init_2,
          t_in, t_track, t_out, t_g);

    hipDeviceReset();
    printf("====> GPU STOP\n");
}
#undef EPS
